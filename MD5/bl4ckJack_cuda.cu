#include "hip/hip_runtime.h"

#include <windows.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "md5_gpu.h"

#define MAX_CHARSET		255		// bytes (non-unicode support atm)
#define MAX_PASSLENGTH	255
#define MAX_PASSCOUNT	1024
__device__ __constant__ char __align__(16) gpu_charset[MAX_CHARSET];
__device__ __constant__ unsigned int gpu_charset_len;

__device__ unsigned long matchCount;
__device__ char __align__(16) matchHashList[MAX_PASSCOUNT][MAX_PASSLENGTH + 1];
__device__ char __align__(16) matchPassList[MAX_PASSCOUNT][MAX_PASSLENGTH + 1];

// init our charset
// init our hashes for matching as well
//! Initialize GPU for Bruteforcing
extern "C" __declspec(dllexport) void bl4ckJackInitGPU(char *charset, int charsetLen) {
	
	if(hipMemcpyToSymbol(HIP_SYMBOL("gpu_charset"), charset, charsetLen+1, 0, hipMemcpyHostToDevice) != hipSuccess) {
		return;
	}

	if(hipMemcpyToSymbol(HIP_SYMBOL("gpu_charset_len"), &charsetLen, sizeof(charsetLen)) != hipSuccess) {
		return;
	}

	if(hipMemset(matchHashList, 0, MAX_PASSCOUNT * MAX_PASSLENGTH) != hipSuccess) {
		return;
	}

	if(hipMemset(matchPassList, 0, MAX_PASSCOUNT * MAX_PASSLENGTH) != hipSuccess) {
		return;
	}

	return;
}

//! Free initialized memory
extern "C" __declspec(dllexport) void bl4ckJackFreeGPU(void) {
	
}

// need to allocate max charset len +1 * index so our kernel can calculate our string
__device__ size_t my_strlen(const char *c) {
	if(!c) return 0;
	register size_t i=0;
	while(c[i]) {
		i++;
	}
	return i;
}

__device__ int my_memcmp ( unsigned char *s1, unsigned char *s2, int n )
{
   int res;
   unsigned char a0;
   unsigned char b0;
   /*
   unsigned char* s1 = (unsigned char*)s1V;
   unsigned char* s2 = (unsigned char*)s2V;
	*/
   while (n != 0) {
      a0 = s1[0];
      b0 = s2[0];
      s1 += 1;
      s2 += 1;
      res = ((int)a0) - ((int)b0);
      if (res != 0)
         return res;
      n -= 1;
   }
   return 0;
}

// kernel will take base value (can calc or assign value to array)
// each thread = compute hash and check btree for result
// each thread will += until its id > stopping key
extern "C" __global__ __declspec(dllexport) void bl4ckJackGenerateGPUInternal(double *start, double *stop, int maxIterations,  char **gpuHashList, int *gpuHashListCount, int *maxSuccess) {
    
    //int index = (blockDim.x * blockIdx.x) + threadIdx.x; //threadIdx.x + blockIdx * blockDim.x;
	//int index = (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	MD5_CTX ctx;
    char input[256]; // max len of our passwd
    unsigned char retBuf[32];
 
    // prime into shared regional memory
    // because i was told this is faster than device memory
    __shared__ char localcharset[256];
    __shared__ int localcharsetLen;

	if(threadIdx.x == 0)
	{
		// load charset/len from gpu mem
		//localcharsetLen=0;
		memcpy(&localcharsetLen, &gpu_charset_len, sizeof(localcharsetLen));
		memcpy(localcharset, gpu_charset, localcharsetLen);
		/*
		while(gpu_charset[localcharsetLen]) {
			localcharset[localcharsetLen] = gpu_charset[localcharsetLen];
			localcharsetLen++;
		}
		*/
		localcharset[localcharsetLen] = '\0';
		
	}

	//Wait for all cache filling to finish
	__syncthreads();
   
    double start_token = *start + index; // base token
    double stop_token = *stop + index;
    double iter = 0;
	int count=0;

    if(stop_token > *stop)
		stop_token -= index;

	if(start_token > *stop) return;
	
	for(iter = start_token; iter <= stop_token; iter += index)
	{

		int base_r = 0;
		int base_iter=0;
		float number = iter - 1;

		
		memset(input, 0, sizeof(input));

		if(number < 0) {
			input[0] = '\0';
		} else {
			do {
				if(base_iter > (sizeof(input)-1)) break;
				base_r = floor(fmod(number, localcharsetLen));
				if(base_r < localcharsetLen)
					input[base_iter++] = localcharset[base_r];
				else
					input[base_iter++] = '=';
				number = floor(number / localcharsetLen) - 1;
			} while(number >= 0);
		}
		input[base_iter] = '\0';
		
		char *p = input;
		char *q = p;
		while(q && *q) ++q;
		for(--q; p < q; ++p, --q)
			*p = *p ^ *q,
			*q = *p ^ *q,
			*p = *p ^ *q;


		// ToBase(iter, input, sizeof(input)-1);
		size_t inputLen = my_strlen(input);
		
		GPUMD5Init(&ctx);
		GPUMD5Update(&ctx, (unsigned char *)input, inputLen);
		GPUMD5Final(&ctx);

		unsigned long ihash=0;
		int match=0;
		
		for(ihash=0; ihash < *gpuHashListCount; ihash++) {
			
			if(!my_memcmp(ctx.digest, (unsigned char *)gpuHashList[ihash], 16)) {
				match=1;
				break;
			}
			
		}

		if(match==1)
		{	
			memcpy(matchHashList[matchCount], retBuf, 16);
			memcpy(matchPassList[matchCount], input, inputLen+1);
			matchCount++;
	
			if(matchCount + 1 > *maxSuccess)
				break;			
		}

		//if(threadIdx.x == 0)
		if(*start < iter)
			*start = iter;

		if(++count > maxIterations)
			break;
	}

    return;
}

// end our init and free all our memory, including btree, etc.

extern "C" __declspec(dllexport) void bl4ckJackGenerateGPU(int block, int thread, int shmem, double *start, double *stop, int maxIterations, char **gpuHashList, int *gpuHashListCount, int *matchCount) {

	//bl4ckJackGenerateGPUInternal<<<block,thread,shmem>>>(start, stop, maxIterations, matchCount);
	bl4ckJackGenerateGPUInternal<<<block,thread>>>(start, stop, maxIterations, gpuHashList, gpuHashListCount, matchCount);
	
	hipDeviceSynchronize();

	if(hipGetLastError() != hipSuccess) {
		OutputDebugString("CUDA Error: ");
		OutputDebugString(hipGetErrorString(hipGetLastError()));
		OutputDebugString("\n");
	}

	// copy success to and from and update passworsd per second
	return;
}


// btree functions
/*
struct node* newNode(struct node* parent, void *data, int dataLen) {
  struct node *node=NULL;
  hipError_t err;
  
  err = hipMalloc((void**)&node, sizeof(struct node));
  if( err != hipSuccess)
	  return NULL;
  
  void *ptr = NULL;

  err = hipMalloc((void **)&ptr, dataLen);
  if( err != hipSuccess)
	  return NULL;

  if(ptr) {
	err = hipMemcpy(ptr, &data, dataLen , hipMemcpyHostToDevice);
	if( err != hipSuccess)
	  return NULL;
  }
  else {
	hipFree(ptr);
	hipFree(node);
	return NULL;
  }
  
  //node->data = ptr;
  err = hipMemcpy(node->data, &ptr, sizeof(ptr) , hipMemcpyHostToDevice);
	if( err != hipSuccess)
	  return NULL;

  //node->dataLen = dataLen;
  err = hipMemcpy(&node->dataLen, &dataLen, sizeof(dataLen) , hipMemcpyHostToDevice);
	if( err != hipSuccess)
	  return NULL;

  //node->left = NULL;
  struct node *n=NULL;
  err = hipMemcpy(&node->left, &n, sizeof(n), hipMemcpyHostToDevice);
	if( err != hipSuccess)
	  return NULL;

  //node->right = NULL;
  err = hipMemcpy(&node->right, &n, sizeof(n) , hipMemcpyHostToDevice);
	if( err != hipSuccess)
	  return NULL;

  //node->parent = parent;
  err = hipMemcpy(&node->parent, &parent, sizeof(parent), hipMemcpyHostToDevice);
	if( err != hipSuccess)
	  return NULL;

  return(node);
}

int lessThan(unsigned char *base, size_t baseLen, unsigned char *compare, size_t compareLen) {


	register unsigned int i=0;
	if(!base) return 0;
	if(!compare) return 0;

	while( i < baseLen && i < compareLen) {
		if(compare[i] > base[i])
			return 0;
		else if(compare[i] == base[i]) {
			i++;
			continue;
		} else
			return 1;
	}

	return 1;
}

__device__ int devicelessThan(unsigned char *base, size_t baseLen, unsigned char *compare, size_t compareLen) {


	register unsigned int i=0;
	if(!base) return 0;
	if(!compare) return 0;

	while( i < baseLen && i < compareLen) {
		if(compare[i] > base[i])
			return 0;
		else if(compare[i] == base[i]) {
			i++;
			continue;
		} else
			return 1;
	}

	return 1;
}
int greaterThan(unsigned char *base, size_t baseLen, unsigned char *compare, size_t compareLen) {
	register unsigned int i=0;
	if(!base) return 0;
	if(!compare) return 0;

	while(i < baseLen && i < compareLen) {
		if(compare[i] < base[i])
			return 0;
		else if(compare[i] == base[i]) {
			i++;
			continue;
		} else
			return 1;
	}
	return 1;
}

int Equals(unsigned char *base, int baseLen, unsigned char *compare, int compareLen) {

	register int i=0;
	if(!base) return 0;
	if(!compare) return 0;

	while(i < baseLen && i < compareLen) {
		if(base[i] != compare[i]) 
			return 0;
		i++;
	}
	return 1;
}

__device__ int deviceEquals(unsigned char *base, int baseLen, unsigned char *compare, int compareLen) {

	register int i=0;
	if(!base) return 0;
	if(!compare) return 0;

	while(i < baseLen && i < compareLen) {
		if(base[i] != compare[i]) 
			return 0;
		i++;
	}
	return 1;
}

__device__ struct node *first_node(struct node *tree) {
    struct node *tmp;
    while(tree){
        tmp = tree;
        tree = tree->left;
    }
    return tmp;
}

__device__ struct node *next_node(struct node *n){

    if(!n) return NULL;

    if(n->right)
        return first_node(n->right);

    while(n->parent && n->parent->right == n)
        n = n->parent;

    if(!n->parent)
        return NULL;

    return n;
}

__device__ int lookup(struct node *parentTree, void *target, int targetLen) {
	struct node *node = first_node(parentTree);
	while(node) {
		if (deviceEquals((unsigned char *)node->data, node->dataLen, (unsigned char *)target, targetLen))
			return 1;
			
		node = next_node(node);
	}
	return 0;
}
*/
/*
__device__ int lookup(struct node* node, void *target, int targetLen) {
  // 1. Base case == empty tree
  // in that case, the target is not found so return false
  if (node == NULL) {
		return 0;
  }
  else {
    // 2. see if found here
    if (deviceEquals((unsigned char *)node->data, node->dataLen, (unsigned char *)target, targetLen))
		return 1;
    else {
      // 3. otherwise recur down the correct subtree
      if (devicelessThan((unsigned char *)target, targetLen, (unsigned char *)node->data, node->dataLen) == 1)
		return(lookup2(node->left, (unsigned char *)target, targetLen));
      else 
		return(lookup2(node->right, (unsigned char *)target, targetLen));
    }
  }
}
*/
/*
struct node* insert(struct node* node, struct node *parentNode, void *data, int dataLen) {
  // 1. If the tree is empty, return a new, single node
  if (node == NULL) {
    return(newNode(parentNode, data, dataLen));
  }
  else {
    // 2. Otherwise, recur down the tree
    if (lessThan((unsigned char *)data, dataLen, (unsigned char *)node->data, node->dataLen) == 1
		|| Equals((unsigned char *)data, dataLen, (unsigned char *)node->data, node->dataLen == 1) )
		node->left = insert(node->left, node, data, dataLen);
    else node->right = insert(node->right, node, data, dataLen);
    return(node); // return the (unchanged) node pointer
  }
}

void destroy(struct node *p)
{
  hipError_t err;
  if (p != 0)
    {
      destroy(p->left);
      destroy(p->right);
	  err = hipFree(p->data);
	  if(err != hipSuccess)
		  return;
      err = hipFree(p);
	  if(err != hipSuccess)
		  return;
    }
}
*/
